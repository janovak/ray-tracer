#include "hip/hip_runtime.h"
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include "color.h"
#include "constants.h"
#include "hittable.h"
#include "hittable_list.h"
#include "sphere.h"
#include "ray.h"

// Camera and viewport constants
__constant__ Point3 d_camera_center;
__constant__ Point3 d_pixel00_loc;
__constant__ Point3 d_pixel_delta_u;
__constant__ Point3 d_pixel_delta_v;

#define GpuErrorCheck(ans) { GpuAssert((ans), __FILE__, __LINE__); }
inline void GpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void InitScene(Hittable **d_list, Hittable **d_world) {
    *(d_list)     = new Sphere(Point3(0, 0, -1), 0.5f);
    *(d_list + 1) = new Sphere(Point3(0, -100.5f, -1), 100);
    *d_world      = new HittableList(d_list, 2);
}

__global__ void FreeScene(Hittable **d_list, Hittable **d_world) {
    delete *(d_list);
    delete *(d_list+1);
    delete *d_world;
}

__device__ Color TraceRay(const Ray& ray, Hittable** world) {
    HitRecord rec;
    if ((*world)->Hit(ray, 0, kInfinity, rec)) {
        return 0.5f*Color(rec.m_normal.X()+1.0f, rec.m_normal.Y()+1.0f, rec.m_normal.Z()+1.0f);
    } else {
        Vec3 unit_direction = UnitVector(ray.Direction());
        float a = 0.5f * (unit_direction.Y() + 1.0f);
        return (1.0f - a) * Color(1.0f, 1.0f, 1.0f) + a * Color(0.5f, 0.7f, 1.0f);
    }
}

__global__ void ProcessImage(Color* d_image, int width, int height, Hittable** d_world) {
    const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx < width && idy < height) {
        Point3 pixel_center = d_pixel00_loc + (idx * d_pixel_delta_u) + (idy * d_pixel_delta_v);
        Vec3 ray_direction = pixel_center - d_camera_center;
        Ray ray(d_camera_center, ray_direction);

        const unsigned int pixel_idx = idy * width + idx;
        d_image[pixel_idx] = TraceRay(ray, d_world);
    }
}

int main() {
    // Initialize the world
    Hittable** d_list;
    Hittable** d_world;

    GpuErrorCheck(hipMalloc((void**)&d_list, 2 * sizeof(Hittable*)));
    GpuErrorCheck(hipMalloc((void**)&d_world, sizeof(Hittable*)));

    InitScene<<<1,1>>>(d_list, d_world);
    GpuErrorCheck(hipDeviceSynchronize());

    // Image dimensions
    const float aspect_ratio = 16.0f / 9.0f;
    const unsigned int image_width = 400;

    // Calculate the image height, and ensure that it's At least 1.
    int image_height = int(image_width / aspect_ratio);
    image_height = (image_height < 1) ? 1 : image_height;

    // Camera

    float focal_length = 1.0f;
    float viewport_height = 2.0f;
    float viewport_width = viewport_height * (float(image_width) / image_height);

    Point3 h_camera_center = Point3(0, 0, 0);
    GpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_camera_center), &h_camera_center, sizeof(Point3)));

    // Calculate the vectors across the horizontal and down the vertical viewport edges.
    Vec3 viewport_u = Vec3(viewport_width, 0, 0);
    Vec3 viewport_v = Vec3(0, -viewport_height, 0);

    // Calculate the horizontal and vertical delta vectors from pixel to pixel.
    Vec3 h_pixel_delta_u = viewport_u / image_width;
    GpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_pixel_delta_u), &h_pixel_delta_u, sizeof(Point3)));
    Vec3 h_pixel_delta_v = viewport_v / image_height;
    GpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_pixel_delta_v), &h_pixel_delta_v, sizeof(Point3)));

    // Calculate the location of the upper left pixel.
    Point3 viewport_upper_left = h_camera_center - Vec3(0, 0, focal_length) - viewport_u / 2 - viewport_v / 2;
    Point3 h_pixel00_loc = viewport_upper_left + 0.5f * (h_pixel_delta_u + h_pixel_delta_v);
    GpuErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_pixel00_loc), &h_pixel00_loc, sizeof(Point3)));

    const unsigned int num_pixels = image_height * image_width;

    // Allocate memory for image on the device
    Color* d_image;
    GpuErrorCheck(hipMalloc((void**)&d_image, num_pixels * sizeof(Color)));

    // Set up grid and block dimensions
    constexpr unsigned int tile_size_x = 8;
    constexpr unsigned int tile_size_y = 8;
    dim3 blocks(image_width / tile_size_x + 1, image_height / tile_size_y + 1);
    dim3 threads(tile_size_x, tile_size_y);

    ProcessImage<<<blocks, threads>>>(d_image, image_width, image_height, d_world);
    GpuErrorCheck(hipDeviceSynchronize());

    // Allocate memory for image on the host
    Color* h_image = (Color*)malloc(num_pixels * sizeof(Color));

    // Copy the result back to the host
    GpuErrorCheck(hipMemcpy(h_image, d_image, num_pixels * sizeof(Color), hipMemcpyDeviceToHost));

    // Output the image
    std::cout << "P3\n" << image_width << ' ' << image_height << "\n255\n";
    for (int j = 0; j < image_height; ++j) {
        for (int i = 0; i < image_width; ++i) {
            int pixel_index = j * image_width + i;
            std::cout << h_image[pixel_index];
        }
    }

    // Free dynamically allocated memory

    // Free GPU memory
    FreeScene<<<1, 1>>>(d_list, d_world);
    GpuErrorCheck(hipDeviceSynchronize());

    hipFree(d_list);
    hipFree(d_world);
    hipFree(d_image);

    // Free host memory
    free(h_image);

    return 0;
}
